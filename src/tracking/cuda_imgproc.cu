#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudawarping.hpp>
#include "tracking/cuda_imgproc.h"
#include "utils/safe_call.h"
#include "math/matrix_type.h"
#include "math/vector_type.h"

namespace fusion
{

FUSION_HOST inline dim3 createGrid(dim3 block, int cols, int rows)
{
    return dim3(div_up(cols, block.x), div_up(rows, block.y));
}

FUSION_DEVICE inline Vector4c renderPoint(
    const Vector3f &point,
    const Vector3f &normal,
    const Vector3f &image,
    const Vector3f &light_pos)
{
    Vector3f colour(4.f / 255.f, 2.f / 255.f, 2.f / 255.f);
    if (!isnan(point.x))
    {
        const float Ka = 0.3f; //ambient coeff
        const float Kd = 0.5f; //diffuse coeff
        const float Ks = 0.2f; //specular coeff
        const float n = 20.f;  //specular power

        const float Ax = image.x; //ambient color,  can be RGB
        const float Dx = image.y; //diffuse color,  can be RGB
        const float Sx = image.z; //specular color, can be RGB
        const float Lx = 1.f;     //light color

        Vector3f L = normalised(light_pos - point);
        Vector3f V = normalised(Vector3f(0.f, 0.f, 0.f) - point);
        Vector3f R = normalised(2 * normal * (normal * L) - L);

        float Ix = Ax * Ka * Dx + Lx * Kd * Dx * fmax(0.f, (normal * L)) + Lx * Ks * Sx * pow(fmax(0.f, (R * V)), n);
        colour = Vector3f(Ix, Ix, Ix);
    }

    return Vector4c(
        static_cast<unsigned char>(__saturatef(colour.x) * 255.f),
        static_cast<unsigned char>(__saturatef(colour.y) * 255.f),
        static_cast<unsigned char>(__saturatef(colour.z) * 255.f),
        255);
}

__global__ void renderSceneK(
    const cv::cuda::PtrStep<Vector4f> vmap,
    const cv::cuda::PtrStep<Vector4f> nmap,
    const Vector3f light_pos,
    cv::cuda::PtrStepSz<Vector4c> dst)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= dst.cols || y >= dst.rows)
        return;

    Vector3f point = ToVector3(vmap.ptr(y)[x]);
    Vector3f normal = ToVector3(nmap.ptr(y)[x]);
    Vector3f pixel(1.f);

    dst.ptr(y)[x] = renderPoint(point, normal, pixel, light_pos);
}

void renderScene(const cv::cuda::GpuMat vmap, const cv::cuda::GpuMat nmap, cv::cuda::GpuMat &image)
{
    dim3 block(8, 8);
    dim3 grid = createGrid(block, vmap.cols, vmap.rows);

    if (image.empty())
        image.create(vmap.rows, vmap.cols, CV_8UC4);

    renderSceneK<<<grid, block>>>(vmap, nmap, Vector3f(5, 5, 5), image);
}

__global__ void renderSceneTexturedK(
    const cv::cuda::PtrStep<Vector4f> vmap,
    const cv::cuda::PtrStep<Vector4f> nmap,
    const cv::cuda::PtrStep<Vector3c> image,
    const Vector3f light_pos,
    cv::cuda::PtrStepSz<Vector4c> dst)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= dst.cols || y >= dst.rows)
        return;

    Vector3f point = ToVector3(vmap.ptr(y)[x]);
    Vector3f normal = ToVector3(nmap.ptr(y)[x]);
    Vector3f pixel = ToVector3f(image.ptr(y)[x]) / 255.f;

    dst.ptr(y)[x] = renderPoint(point, normal, pixel, light_pos);
}

void renderSceneTextured(const cv::cuda::GpuMat vmap, const cv::cuda::GpuMat nmap, const cv::cuda::GpuMat image, cv::cuda::GpuMat &out)
{
    dim3 block(8, 8);
    dim3 grid = createGrid(block, vmap.cols, vmap.rows);

    if (out.empty())
        out.create(vmap.rows, vmap.cols, CV_8UC4);

    renderSceneTexturedK<<<grid, block>>>(vmap, nmap, image, Vector3f(5, 5, 5), out);
}

__global__ void ToSemiDenseImageK(
    const cv::cuda::PtrStepSz<float> image,
    const cv::cuda::PtrStepSz<float> intensity_dx,
    const cv::cuda::PtrStepSz<float> intensity_dy,
    cv::cuda::PtrStepSz<float> semi,
    float th_dx, float th_dy)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= image.cols || y >= image.rows)
        return;

    semi.ptr(y)[x] = 255;

    auto dx = intensity_dx.ptr(y)[x];
    auto dy = intensity_dy.ptr(y)[x];

    if (dx > th_dx || dy > th_dy)
    {
        semi.ptr(y)[x] = image.ptr(y)[x];
    }
}

void convert_image_to_semi_dense(const cv::cuda::GpuMat image, const cv::cuda::GpuMat dx, const cv::cuda::GpuMat dy, cv::cuda::GpuMat &semi, float th_dx, float th_dy)
{
    if (semi.empty())
        semi.create(image.size(), image.type());

    dim3 block(8, 4);
    dim3 grid = createGrid(block, image.cols, image.rows);

    ToSemiDenseImageK<<<grid, block>>>(image, dx, dy, semi, th_dx, th_dy);
}

void build_semi_dense_pyramid(const std::vector<cv::cuda::GpuMat> image_pyr, const std::vector<cv::cuda::GpuMat> dx_pyr, const std::vector<cv::cuda::GpuMat> dy_pyr, std::vector<cv::cuda::GpuMat> &semi_pyr, float th_dx, float th_dy)
{
    if (semi_pyr.size() != image_pyr.size())
        semi_pyr.resize(image_pyr.size());

    for (int level = 0; level < image_pyr.size(); ++level)
    {
        convert_image_to_semi_dense(image_pyr[level], dx_pyr[level], dy_pyr[level], semi_pyr[level], th_dx, th_dy);
    }
}

FUSION_DEVICE inline Vector3c interpolate_bilinear(const cv::cuda::PtrStepSz<Vector3c> image, float x, float y)
{
    int u = std::floor(x), v = std::floor(y);
    float coeff_x = x - (float)u, coeff_y = y - (float)v;
    Vector3f result = ToVector3f((image.ptr(v)[u] * (1 - coeff_x) + image.ptr(v)[u + 1] * coeff_x) * (1 - coeff_y) +
                                 (image.ptr(v + 1)[u] * (1 - coeff_x) + image.ptr(v + 1)[u + 1] * coeff_x) * coeff_y);
    return ToVector3c(result);
}

__global__ void warp_image_kernel(const cv::cuda::PtrStepSz<Vector3c> src,
                                  const cv::cuda::PtrStep<Vector4f> vmap_dst,
                                  const Matrix3x4f pose,
                                  const Eigen::Matrix3f K,
                                  cv::cuda::PtrStep<Vector3c> dst)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= src.cols || y >= src.rows)
        return;

    dst.ptr(y)[x] = Vector3c(0);
    Vector3f dst_pt_src = pose(ToVector3(vmap_dst.ptr(y)[x]));

    float u = K(0,0) * dst_pt_src.x / dst_pt_src.z + K(0,2);
    float v = K(1,1) * dst_pt_src.y / dst_pt_src.z + K(1,2);
    if (u >= 1 && v >= 1 && u < src.cols - 1 && v < src.rows - 1)
    {
        dst.ptr(y)[x] = interpolate_bilinear(src, u, v);
    }
}

void warp_image(const cv::cuda::GpuMat src, const cv::cuda::GpuMat vmap_dst, const Sophus::SE3d pose, const Eigen::Matrix3f K, cv::cuda::GpuMat &dst)
{
    if (dst.empty())
        dst.create(src.size(), src.type());

    dim3 block(8, 4);
    dim3 grid = createGrid(block, src.cols, src.rows);

    warp_image_kernel<<<grid, block>>>(src, vmap_dst, pose.cast<float>().matrix3x4(), K, dst);
}

FUSION_HOST void filterDepthBilateral(const cv::cuda::GpuMat src, cv::cuda::GpuMat &dst)
{
    cv::cuda::bilateralFilter(src, dst, 5, 1, 1);
}

FUSION_HOST void pyrDownDepth(const cv::cuda::GpuMat src, cv::cuda::GpuMat &dst)
{
    cv::cuda::resize(src, dst, cv::Size(0, 0), 0.5, 0.5);
}

FUSION_HOST void pyrDownImage(const cv::cuda::GpuMat src, cv::cuda::GpuMat &dst)
{
    cv::cuda::pyrDown(src, dst);
}

FUSION_HOST void pyrDownVMap(const cv::cuda::GpuMat src, cv::cuda::GpuMat &dst)
{
    cv::cuda::resize(src, dst, cv::Size(0, 0), 0.5, 0.5);
}

__global__ void computeDerivativeK(
    cv::cuda::PtrStepSz<float> image,
    cv::cuda::PtrStep<float> dx,
    cv::cuda::PtrStep<float> dy)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= image.cols - 1 || y >= image.rows - 1)
        return;

    int x10 = max(x - 1, 0);
    int x01 = min(x + 1, image.cols);
    int y10 = max(y - 1, 0);
    int y01 = min(y + 1, image.rows);

    dx.ptr(y)[x] = (image.ptr(y)[x01] - image.ptr(y)[x10]) * 0.5;
    dy.ptr(y)[x] = (image.ptr(y01)[x] - image.ptr(y10)[x]) * 0.5;
}

FUSION_HOST void computeDerivative(const cv::cuda::GpuMat image, cv::cuda::GpuMat &dx, cv::cuda::GpuMat &dy)
{
    if (dx.empty())
        dx.create(image.size(), image.type());
    if (dy.empty())
        dy.create(image.size(), image.type());

    dim3 block(8, 8);
    dim3 grid(div_up(image.cols, block.x), div_up(image.rows, block.y));

    computeDerivativeK<<<grid, block>>>(image, dx, dy);
}

__global__ void backProjectDepthK(const cv::cuda::PtrStepSz<float> depth, cv::cuda::PtrStep<Vector4f> vmap, Eigen::Matrix3f KInv)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x > depth.cols - 1 || y > depth.rows - 1)
        return;

    vmap.ptr(y)[x] = Vector4f(nanf("NAN"), nanf("NAN"), nanf("NAN"), -1.0f);
    float z = depth.ptr(y)[x];
    // z = (z == z) ? z : nanf("NAN");
    if (z > 0.3f && z < 5.0f)
    {
        vmap.ptr(y)[x] = Vector4f(z * (x - KInv(0,2)) * KInv(0,0), z * (y - KInv(1,2)) * KInv(1,1), z, 1.0f);
    }
}

FUSION_HOST void backProjectDepth(const cv::cuda::GpuMat depth, cv::cuda::GpuMat &vmap, const Eigen::Matrix3f &KInv)
{
    if (vmap.empty())
        vmap.create(depth.size(), CV_32FC4);

    dim3 block(8, 8);
    dim3 grid = createGrid(block, depth.cols, depth.rows);

    backProjectDepthK<<<grid, block>>>(depth, vmap, KInv);
}

__global__ void computeNMapK(cv::cuda::PtrStepSz<Vector4f> vmap, cv::cuda::PtrStep<Vector4f> nmap)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= vmap.cols - 1 || y >= vmap.rows - 1)
        return;

    int x10 = max(x - 1, 0);
    int x01 = min(x + 1, vmap.cols);
    int y10 = max(y - 1, 0);
    int y01 = min(y + 1, vmap.rows);

    Vector3f v00 = ToVector3(vmap.ptr(y)[x10]);
    Vector3f v01 = ToVector3(vmap.ptr(y)[x01]);
    Vector3f v10 = ToVector3(vmap.ptr(y10)[x]);
    Vector3f v11 = ToVector3(vmap.ptr(y01)[x]);

    nmap.ptr(y)[x] = Vector4f(normalised((v01 - v00).cross(v11 - v10)), 1.f);
}

FUSION_HOST void computeNMap(const cv::cuda::GpuMat vmap, cv::cuda::GpuMat &nmap)
{
    if (nmap.empty())
        nmap.create(vmap.size(), vmap.type());

    dim3 block(8, 8);
    dim3 grid = createGrid(block, vmap.cols, vmap.rows);

    computeNMapK<<<grid, block>>>(vmap, nmap);
}

__global__ void NVmapToEdgeKernel(cv::cuda::PtrStepSz<Vector4f> normal, cv::cuda::PtrStepSz<Vector4f> vertex,
                                  cv::cuda::PtrStepSz<unsigned char> edge, 
                                  float lamb, float tao, int win_size, int step)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    // need to assign boundary values to zero first???
    if(x < step || x >= normal.cols-step || y < step || y >= normal.rows-step){
        edge.ptr(y)[x] = 0;
        return;
    }

    float sum_norm = 0.;
    float sum_vert = 0.;
    Vector4f n = normal.ptr(y)[x];
    Vector4f v = vertex.ptr(y)[x];
    float phi_d = -100.;
    float phi_c = -100.;
    for(int idx=0; idx<win_size*win_size; idx++){
        int c = x-step+(idx/win_size);
        int r = y-step+(idx%win_size);

        Vector4f n_cur = normal.ptr(r)[c];
        Vector4f v_cur = vertex.ptr(r)[c];

        sum_norm += (n_cur.x + n_cur.y + n_cur.z + n_cur.w);
        sum_vert += (v_cur.x + v_cur.y + v_cur.z + v_cur.w);
        if(isnan(sum_norm) || isnan(sum_vert)){
            edge.ptr(y)[x] = 0;
            return;
        }

        // depth term
        float depth_term = (v_cur-v)* n;
        phi_d = abs(depth_term) > phi_d ? abs(depth_term) : phi_d;

        // convex term
        // float convex_term = 1 - n_cur * n;
        float convex_term = depth_term >= 0 ? (1- n_cur*n) : 0.; 
        phi_c = convex_term > phi_c ? convex_term : phi_c;

    }

    // calculate edge value
    // uchar e = short((phi_d+lamb*phi_c)<tao)*255;
    edge.ptr(y)[x] = short((phi_d+lamb*phi_c)<tao)*255;
}

FUSION_HOST void NVmapToEdge(const cv::cuda::GpuMat normal, const cv::cuda::GpuMat vertex, 
                             cv::cuda::GpuMat &edge, 
                             float lamb, float tao, int win_size, int step)
{
    if(edge.empty())
        edge.create(normal.size(), CV_8UC1);
    // set dimensions
    dim3 thread(8, 8);
    dim3 block = createGrid(thread, normal.cols, normal.rows);

    NVmapToEdgeKernel<<<block, thread>>>(normal, vertex, edge, lamb, tao, win_size, step);
    hipDeviceSynchronize();
    safe_call(hipGetLastError());
}

} // namespace fusion
