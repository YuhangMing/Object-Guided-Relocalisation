#include "hip/hip_runtime.h"
#include "voxel_hashing/map_proc.h"
#include "math/matrix_type.h"
#include "math/vector_type.h"
#include "utils/safe_call.h"
#include "macros.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudaarithm.hpp>
#include <thrust/device_vector.h>

namespace fusion
{
namespace cuda
{

FUSION_DEVICE inline bool is_vertex_visible(
    Vector3f pt, Matrix3x4f inv_pose,
    int cols, int rows, float fx,
    float fy, float cx, float cy)
{
    pt = inv_pose(pt);
    Vector2f pt2d = Vector2f(fx * pt.x / pt.z + cx, fy * pt.y / pt.z + cy);
    return !(pt2d.x < 0 || pt2d.y < 0 ||
             pt2d.x > cols - 1 || pt2d.y > rows - 1 ||
             pt.z < param.zmin_update || pt.z > param.zmax_update);
}

FUSION_DEVICE inline bool is_block_visible(
    const Vector3i &block_pos,
    Matrix3x4f inv_pose,
    int cols, int rows, float fx,
    float fy, float cx, float cy)
{
    float scale = param.block_size_metric();
#pragma unroll
    for (int corner = 0; corner < 8; ++corner)
    {
        Vector3i tmp = block_pos;
        tmp.x += (corner & 1) ? 1 : 0;
        tmp.y += (corner & 2) ? 1 : 0;
        tmp.z += (corner & 4) ? 1 : 0;

        if (is_vertex_visible(tmp * scale, inv_pose, cols, rows, fx, fy, cx, cy))
            return true;
    }

    return false;
}

__global__ void check_visibility_flag_kernel(
    MapStorage map_struct, uchar *flag, Matrix3x4f inv_pose,
    int cols, int rows, float fx, float fy, float cx, float cy)
{
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= param.num_total_hash_entries_)
        return;

    HashEntry &current = map_struct.hash_table_[idx];
    if (current.ptr_ != -1)
    {
        switch (flag[idx])
        {
        default:
        {
            if (is_block_visible(current.pos_, inv_pose, cols, rows, fx, fy, cx, cy))
            {
                flag[idx] = 1;
            }
            else
            {
                // map_struct.delete_block(current);
                flag[idx] = 0;
            }

            return;
        }
        case 2:
        {
            flag[idx] = 1;
            return;
        }
        }
    }
}

__global__ void copy_visible_block_kernel(HashEntry *hash_table, HashEntry *visible_block, const uchar *flag, const int *pos)
{
    // printf("Block %d thread %d, ", blockIdx.x, threadIdx.x);

    
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= param.num_total_hash_entries_)
        return;

    if (flag[idx] == 1 && pos[idx] < param.num_total_hash_entries_)
        visible_block[pos[idx]] = hash_table[idx];
}

FUSION_DEVICE inline Vector2f project(
    Vector3f pt, float fx, float fy, float cx, float cy)
{
    return Vector2f(fx * pt.x / pt.z + cx, fy * pt.y / pt.z + cy);
}

FUSION_DEVICE inline Vector3f unproject(
    int x, int y, float z, float invfx, float invfy, float cx, float cy)
{
    return Vector3f(invfx * (x - cx) * z, invfy * (y - cy) * z, z);
}

FUSION_DEVICE inline Vector3f unproject_world(
    int x, int y, float z, float invfx,
    float invfy, float cx, float cy, Matrix3x4f pose)
{
    return pose(unproject(x, y, z, invfx, invfy, cx, cy));
}

FUSION_DEVICE inline int create_block(MapStorage &map_struct, const Vector3i block_pos)
{
    int hash_index;
    createBlock(map_struct, block_pos, hash_index);
    return hash_index;
}

__global__ void create_blocks_kernel(MapStorage map_struct, cv::cuda::PtrStepSz<float> depth,
                                     float invfx, float invfy, float cx, float cy,
                                     Matrix3x4f pose, uchar *flag)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= depth.cols || y >= depth.rows)
        return;

    float z = depth.ptr(y)[x];
    if (isnan(z) || z < param.zmin_update || z > param.zmax_update)
        return;

    float z_thresh = param.truncation_dist() * 0.5;
    float z_near = max(param.zmin_update, z - z_thresh);
    float z_far = min(param.zmax_update, z + z_thresh);
    if (z_near >= z_far)
        return;

    Vector3i block_near = voxelPosToBlockPos(worldPtToVoxelPos(unproject_world(x, y, z_near, invfx, invfy, cx, cy, pose), param.voxel_size));
    Vector3i block_far = voxelPosToBlockPos(worldPtToVoxelPos(unproject_world(x, y, z_far, invfx, invfy, cx, cy, pose), param.voxel_size));

    Vector3i d = block_far - block_near;
    Vector3i increment = Vector3i(d.x < 0 ? -1 : 1, d.y < 0 ? -1 : 1, d.z < 0 ? -1 : 1);
    Vector3i incre_abs = Vector3i(abs(d.x), abs(d.y), abs(d.z));
    Vector3i incre_err = Vector3i(incre_abs.x << 1, incre_abs.y << 1, incre_abs.z << 1);

    int err_1;
    int err_2;

    // Bresenham's line algorithm
    // details see : https://en.m.wikipedia.org/wiki/Bresenham%27s_line_algorithm
    if ((incre_abs.x >= incre_abs.y) && (incre_abs.x >= incre_abs.z))
    {
        err_1 = incre_err.y - 1;
        err_2 = incre_err.z - 1;
        flag[create_block(map_struct, block_near)] = 2;
        for (int i = 0; i < incre_abs.x; ++i)
        {
            if (err_1 > 0)
            {
                block_near.y += increment.y;
                err_1 -= incre_err.x;
            }

            if (err_2 > 0)
            {
                block_near.z += increment.z;
                err_2 -= incre_err.x;
            }

            err_1 += incre_err.y;
            err_2 += incre_err.z;
            block_near.x += increment.x;
            flag[create_block(map_struct, block_near)] = 2;
        }
    }
    else if ((incre_abs.y >= incre_abs.x) && (incre_abs.y >= incre_abs.z))
    {
        err_1 = incre_err.x - 1;
        err_2 = incre_err.z - 1;
        flag[create_block(map_struct, block_near)] = 2;
        for (int i = 0; i < incre_abs.y; ++i)
        {
            if (err_1 > 0)
            {
                block_near.x += increment.x;
                err_1 -= incre_err.y;
            }

            if (err_2 > 0)
            {
                block_near.z += increment.z;
                err_2 -= incre_err.y;
            }

            err_1 += incre_err.x;
            err_2 += incre_err.z;
            block_near.y += increment.y;
            flag[create_block(map_struct, block_near)] = 2;
        }
    }
    else
    {
        err_1 = incre_err.y - 1;
        err_2 = incre_err.x - 1;
        flag[create_block(map_struct, block_near)] = 2;
        for (int i = 0; i < incre_abs.z; ++i)
        {
            if (err_1 > 0)
            {
                block_near.y += increment.y;
                err_1 -= incre_err.z;
            }

            if (err_2 > 0)
            {
                block_near.x += increment.x;
                err_2 -= incre_err.z;
            }

            err_1 += incre_err.y;
            err_2 += incre_err.x;
            block_near.z += increment.z;
            flag[create_block(map_struct, block_near)] = 2;
        }
    }
}

__global__ void update_map_kernel(MapStorage map_struct,
                                  HashEntry *visible_blocks,
                                  uint count_visible_block,
                                  cv::cuda::PtrStepSz<float> depth,
                                  Matrix3x4f inv_pose,
                                  float fx, float fy,
                                  float cx, float cy)
{
    if (blockIdx.x >= param.num_total_hash_entries_ || blockIdx.x >= count_visible_block)
        return;

    HashEntry &current = visible_blocks[blockIdx.x];

    Vector3i voxel_pos = blockPosToVoxelPos(current.pos_);
    float dist_thresh = param.truncation_dist();
    float inv_dist_thresh = 1.0 / dist_thresh;

#pragma unroll
    for (int block_idx_z = 0; block_idx_z < 8; ++block_idx_z)
    {
        Vector3i local_pos = Vector3i(threadIdx.x, threadIdx.y, block_idx_z);
        Vector3f pt = inv_pose(voxelPosToWorldPt(voxel_pos + local_pos, param.voxel_size));

        int u = __float2int_rd(fx * pt.x / pt.z + cx + 0.5);
        int v = __float2int_rd(fy * pt.y / pt.z + cy + 0.5);
        if (u < 0 || v < 0 || u > depth.cols - 1 || v > depth.rows - 1)
            continue;

        float dist = depth.ptr(v)[u];
        if (isnan(dist) || dist < 1e-2 || dist > param.zmax_update || dist < param.zmin_update)
            continue;

        float sdf = dist - pt.z;
        if (sdf < -dist_thresh)
            continue;

        sdf = fmin(1.0f, sdf * inv_dist_thresh);
        const int local_idx = localPosToLocalIdx(local_pos);
        Voxel &voxel = map_struct.voxels_[current.ptr_ + local_idx];

        auto sdf_p = voxel.getSDF();
        auto weight_p = voxel.getWeight();
        auto weight = 1 / (dist);

        if (weight_p < 1e-3)
        {
            voxel.setSDF(sdf);
            voxel.setWeight(weight);
            continue;
        }

        // fuse depth
        sdf_p = (sdf_p * weight_p + sdf * weight) / (weight_p + weight);
        voxel.setSDF(sdf_p);
        voxel.setWeight(weight_p + weight);
    }
}

__global__ void update_map_with_colour_kernel(MapStorage map_struct,
                                              HashEntry *visible_blocks,
                                              uint count_visible_block,
                                              cv::cuda::PtrStepSz<float> depth,
                                              cv::cuda::PtrStepSz<Vector3c> image,
                                              Matrix3x4f inv_pose,
                                              float fx, float fy,
                                              float cx, float cy)
{
    if (blockIdx.x >= param.num_total_hash_entries_ || blockIdx.x >= count_visible_block)
        return;

    HashEntry &current = visible_blocks[blockIdx.x];

    Vector3i voxel_pos = blockPosToVoxelPos(current.pos_);
    float dist_thresh = param.truncation_dist();
    float inv_dist_thresh = 1.0 / dist_thresh;

#pragma unroll
    for (int block_idx_z = 0; block_idx_z < 8; ++block_idx_z)
    {
        Vector3i local_pos = Vector3i(threadIdx.x, threadIdx.y, block_idx_z);
        Vector3f pt = inv_pose(voxelPosToWorldPt(voxel_pos + local_pos, param.voxel_size));

        int u = __float2int_rd(fx * pt.x / pt.z + cx + 0.5);
        int v = __float2int_rd(fy * pt.y / pt.z + cy + 0.5);
        if (u < 0 || v < 0 || u > depth.cols - 1 || v > depth.rows - 1)
            continue;

        float dist = depth.ptr(v)[u];
        if (isnan(dist) || dist < 1e-2 || dist > param.zmax_update || dist < param.zmin_update)
            continue;

        float sdf = dist - pt.z;
        if (sdf < -dist_thresh)
            continue;

        sdf = fmin(1.0f, sdf * inv_dist_thresh);
        const int local_idx = localPosToLocalIdx(local_pos);
        Voxel &voxel = map_struct.voxels_[current.ptr_ + local_idx];

        auto sdf_p = voxel.getSDF();
        auto weight_p = voxel.getWeight();
        auto weight = 1 / (dist * dist);

        // update colour
        auto colour_new = image.ptr(v)[u];
        auto colour_p = voxel.rgb;

        if (voxel.weight == 0)
        {
            // printf("Initialising map, sdf = %d. \n", sdf);
            voxel.setSDF(sdf);
            voxel.setWeight(weight);
            voxel.rgb = colour_new;
            continue;
        }

        // printf("Fusing depth and color in map. \n");
        // fuse depth
        sdf_p = (sdf_p * weight_p + sdf * weight) / (weight_p + weight);
        voxel.setSDF(sdf_p);
        voxel.setWeight(weight_p + weight);

        // fuse colour
        colour_p = ToVector3c((colour_p * weight_p + colour_new * weight) / (weight_p + weight));
        voxel.rgb = colour_p;
    }
}

__global__ void update_map_weighted_kernel(
    MapStorage map_struct,
    HashEntry *visible_blocks,
    uint count_visible_block,
    cv::cuda::PtrStepSz<float> depth,
    cv::cuda::PtrStepSz<Vector4f> normal,
    cv::cuda::PtrStepSz<Vector3c> image,
    Matrix3x4f inv_pose,
    float fx, float fy,
    float cx, float cy)
{
    if (blockIdx.x >= param.num_total_hash_entries_ || blockIdx.x >= count_visible_block)
        return;

    HashEntry &current = visible_blocks[blockIdx.x];

    if (current.ptr_ < 0)
        return;

    Vector3i voxel_pos = blockPosToVoxelPos(current.pos_);
    float dist_thresh = param.truncation_dist();
    float inv_dist_thresh = 1.0 / dist_thresh;

    #pragma unroll
    for (int block_idx_z = 0; block_idx_z < 8; ++block_idx_z)
    {
        Vector3i local_pos = Vector3i(threadIdx.x, threadIdx.y, block_idx_z);
        Vector3f pt = inv_pose(voxelPosToWorldPt(voxel_pos + local_pos, param.voxel_size));

        int u = __float2int_rd(fx * pt.x / pt.z + cx + 0.5);
        int v = __float2int_rd(fy * pt.y / pt.z + cy + 0.5);
        if (u < 0 || v < 0 || u > depth.cols - 1 || v > depth.rows - 1)
            continue;

        float dist = depth.ptr(v)[u];
        auto n_c = ToVector3(normal.ptr(v)[u]);
        if (isnan(dist) || isnan(n_c.x) || dist > param.zmax_update || dist < param.zmin_update)
            continue;

        float sdf = dist - pt.z;
        if (sdf < -dist_thresh)
            continue;

        sdf = fmin(1.0f, sdf * inv_dist_thresh);
        const int local_idx = localPosToLocalIdx(local_pos);
        Voxel &voxel = map_struct.voxels_[current.ptr_ + local_idx];

        auto sdf_p = voxel.getSDF();
        auto weight_p = voxel.getWeight();
        auto weight = abs(sin(n_c.z)) / (dist * dist);

        // update colour
        auto colour_new = image.ptr(v)[u];
        auto colour_p = voxel.rgb;

        if (voxel.weight == 0)
        {
            voxel.setSDF(sdf);
            voxel.setWeight(weight);
            voxel.rgb = colour_new;
            continue;
        }

        // fuse depth
        sdf_p = (sdf_p * weight_p + sdf * weight) / (weight_p + weight);
        voxel.setSDF(sdf_p);
        voxel.setWeight(weight_p + weight);

        // fuse colour
        if(voxel.label == 0){
            colour_p = ToVector3c((colour_p * weight_p + colour_new * weight) / (weight_p + weight));
            voxel.rgb = colour_p;
        }
    }
}

__global__ void update_map_with_object_kernel(
    MapStorage map_struct,
    HashEntry *visible_blocks,
    uint count_visible_block,
    cv::cuda::PtrStepSz<float> depth,
    cv::cuda::PtrStepSz<Vector3c> image,
    cv::cuda::PtrStepSz<unsigned char> mask,
    Matrix3x4f inv_pose,
    float fx, float fy,
    float cx, float cy)
{
    // current block is a valid entry & inside the visible block
    if (blockIdx.x >= param.num_total_hash_entries_ || blockIdx.x >= count_visible_block)
        return;

    // get current block
    HashEntry &current = visible_blocks[blockIdx.x];

    // check if current block is assigned
    if (current.ptr_ < 0)
        return;

    // get voxel start position and thresh info 
    Vector3i voxel_pos = blockPosToVoxelPos(current.pos_);
    float dist_thresh = param.truncation_dist();

    // loop through all voxels (8, only along z-axis)
    int confident_thre = 2;
    #pragma unroll
    for (int block_idx_z = 0; block_idx_z < 8; ++block_idx_z)
    {
        Vector3i local_pos = Vector3i(threadIdx.x, threadIdx.y, block_idx_z);
        Vector3f pt = inv_pose(voxelPosToWorldPt(voxel_pos + local_pos, param.voxel_size));

        // get pixel coordinates
        int u = __float2int_rd(fx * pt.x / pt.z + cx + 0.5);
        int v = __float2int_rd(fy * pt.y / pt.z + cy + 0.5);
        // check if it falls in the image range
        if (u < 0 || v < 0 || u > depth.cols - 1 || v > depth.rows - 1)
            continue;

        // get object label
        auto label = mask.ptr(v)[u];
        if(label == 0)
            continue;

        // get depth value
        float dist = depth.ptr(v)[u];
        // check if it is in range
        if (isnan(dist) || dist > param.zmax_update || dist < param.zmin_update)
            continue;

        // check if sdf in the truncated range
        float sdf = dist - pt.z;
        if (sdf < -dist_thresh)
            continue;

        const int local_idx = localPosToLocalIdx(local_pos);
        Voxel &voxel = map_struct.voxels_[current.ptr_ + local_idx];

        // DO NOT create new voxels here !!
        auto weight_p = voxel.getWeight();;
        if(weight_p == 0)
            continue;

        // UPDATE object info
        // Vector3c color_new(int(33554431 * int(label) % 255), 
        //                    int(32767 * int(label) % 255),
        //                    int(2097151 * int(label) % 255));
        // the label is consistent
        if(label == voxel.label)
        {
            voxel.count = min(255, voxel.count+1);
        }
        // the label is Inconsistent
        else
        {
            // check with backup label
            if(label == voxel.label_backup)
            {
                voxel.count_backup = min(255, voxel.count_backup+1);
                // voxel.rgb_backup = color_new;
            }
            // update back_up label if the previous one is detected few times only
            // discard new detection if previous one is already very confident
            else
            {
                if(voxel.count_backup <= confident_thre)
                {
                    voxel.label_backup = label;
                    voxel.count_backup = 1;
                    // voxel.rgb_backup = color_new;
                }
            }

            // compare count
            // switch if 1) prev is bg, object detected is confident enough
            //           2) prev is ob, has more count
            if ( (voxel.count < voxel.count_backup && voxel.label !=0) ||
                 (voxel.count_backup > confident_thre && voxel.label ==0) )
            {
                short tmp = voxel.label;
                voxel.label = voxel.label_backup;
                voxel.label_backup = tmp;

                tmp = voxel.count;
                voxel.count = voxel.count_backup;
                voxel.count_backup = tmp;

                // Vector3c tmpC = voxel.rgb;
                // voxel.rgb = voxel.rgb_backup;
                // voxel.rgb_backup = tmpC;
            }
        }
    } // end block_idx_z
}

void update(
    MapStorage map_struct,
    MapState state,
    const cv::cuda::GpuMat depth,
    const cv::cuda::GpuMat image,
    const Sophus::SE3d &frame_pose,
    const Eigen::Matrix3f K,
    cv::cuda::GpuMat &cv_flag,
    cv::cuda::GpuMat &cv_pos_array,
    HashEntry *visible_blocks,
    uint &visible_block_count)
{
    if (cv_flag.empty())
        cv_flag.create(1, state.num_total_hash_entries_, CV_8UC1);
    if (cv_pos_array.empty())
        cv_pos_array.create(1, state.num_total_hash_entries_, CV_32SC1);

    thrust::device_ptr<uchar> flag(cv_flag.ptr<uchar>());
    thrust::device_ptr<int> pos_array(cv_pos_array.ptr<int>());

    const int cols = depth.cols;
    const int rows = depth.rows;

    dim3 thread(8, 8);
    dim3 block(div_up(cols, thread.x), div_up(rows, thread.y));

    create_blocks_kernel<<<block, thread>>>(
        map_struct,
        depth,
        1.0/K(0,0),
        1.0/K(1,1),
        K(0,2), K(1,2),
        frame_pose.cast<float>().matrix3x4(),
        flag.get());

    thread = dim3(MAX_THREAD);
    block = dim3(div_up(state.num_total_hash_entries_, thread.x));

    check_visibility_flag_kernel<<<block, thread>>>(
        map_struct,
        flag.get(),
        frame_pose.inverse().cast<float>().matrix3x4(),
        cols, rows,
        K(0,0), K(1,1),
        K(0,2), K(1,2));

    thrust::exclusive_scan(flag, flag + state.num_total_hash_entries_, pos_array);

    copy_visible_block_kernel<<<block, thread>>>(
        map_struct.hash_table_,
        visible_blocks,
        flag.get(),
        pos_array.get());

    visible_block_count = pos_array[state.num_total_hash_entries_ - 1];
    
    printf("visible_block_count = %d\n", visible_block_count);
    if (visible_block_count == 0)
        return;

    thread = dim3(8, 8);
    block = dim3(visible_block_count);

    update_map_with_colour_kernel<<<block, thread>>>(
        map_struct,
        visible_blocks,
        visible_block_count,
        depth, image,
        frame_pose.inverse().cast<float>().matrix3x4(),
        K(0,0), K(1,1),
        K(0,2), K(1,2));
}

// MAIN FUSING FUNCTION
void update_weighted(
    MapStorage map_struct,
    MapState state,
    const cv::cuda::GpuMat depth,
    const cv::cuda::GpuMat normal,
    const cv::cuda::GpuMat image,
    const Sophus::SE3d &frame_pose,
    const Eigen::Matrix3f K,
    cv::cuda::GpuMat &cv_flag,
    cv::cuda::GpuMat &cv_pos_array,
    HashEntry *visible_blocks,
    uint &visible_block_count)
{
    if (cv_flag.empty())
        cv_flag.create(1, state.num_total_hash_entries_, CV_8UC1);
    if (cv_pos_array.empty())
        cv_pos_array.create(1, state.num_total_hash_entries_, CV_32SC1);

    thrust::device_ptr<uchar> flag(cv_flag.ptr<uchar>());
    thrust::device_ptr<int> pos_array(cv_pos_array.ptr<int>());

    const int cols = depth.cols;
    const int rows = depth.rows;

    dim3 thread(8, 8);
    dim3 block(div_up(cols, thread.x), div_up(rows, thread.y));

    create_blocks_kernel<<<block, thread>>>(
        map_struct,
        depth,
        1.0/K(0,0),
        1.0/K(1,1),
        K(0,2), K(1,2),
        frame_pose.cast<float>().matrix3x4(),
        flag.get());
    hipDeviceSynchronize();
    safe_call(hipGetLastError());

    thread = dim3(MAX_THREAD);
    block = dim3(div_up(state.num_total_hash_entries_, thread.x));

    check_visibility_flag_kernel<<<block, thread>>>(
        map_struct,
        flag.get(),
        frame_pose.inverse().cast<float>().matrix3x4(),
        cols, rows,
        K(0,0), K(1,1),
        K(0,2), K(1,2));
    hipDeviceSynchronize();
    safe_call(hipGetLastError());

    thrust::exclusive_scan(flag, flag + state.num_total_hash_entries_, pos_array);

    copy_visible_block_kernel<<<block, thread>>>(
        map_struct.hash_table_,
        visible_blocks,
        flag.get(),
        pos_array.get());
    hipDeviceSynchronize();
    safe_call(hipGetLastError());

    visible_block_count = pos_array[state.num_total_hash_entries_ - 1];

    if (visible_block_count == 0)
        return;

    thread = dim3(8, 8);
    block = dim3(visible_block_count);

    update_map_weighted_kernel<<<block, thread>>>(
        map_struct,
        visible_blocks,
        visible_block_count,
        depth,
        normal,
        image,
        frame_pose.inverse().cast<float>().matrix3x4(),
        K(0,0), K(1,1),
        K(0,2), K(1,2));
    hipDeviceSynchronize();
    safe_call(hipGetLastError());
}

void create_new_block(
    MapStorage map_struct,
    MapState state,
    const cv::cuda::GpuMat depth,
    const Sophus::SE3d &frame_pose,
    const Eigen::Matrix3f K,
    cv::cuda::GpuMat &cv_flag,
    cv::cuda::GpuMat &cv_pos_array,
    HashEntry *visible_blocks,
    uint &visible_block_count)
{
    if (cv_flag.empty())
        cv_flag.create(1, state.num_total_hash_entries_, CV_8UC1);
    if (cv_pos_array.empty())
        cv_pos_array.create(1, state.num_total_hash_entries_, CV_32SC1);

    thrust::device_ptr<uchar> flag(cv_flag.ptr<uchar>());
    thrust::device_ptr<int> pos_array(cv_pos_array.ptr<int>());

    const int cols = depth.cols;
    const int rows = depth.rows;

    dim3 thread(8, 8);
    dim3 block(div_up(cols, thread.x), div_up(rows, thread.y));

    create_blocks_kernel<<<block, thread>>>(
        map_struct,
        depth,
        1.0/K(0,0),
        1.0/K(1,1),
        K(0,2), K(1,2),
        frame_pose.cast<float>().matrix3x4(),
        flag.get());
    hipDeviceSynchronize();
    safe_call(hipGetLastError());
    printf("created blocks   ");

    thread = dim3(MAX_THREAD);
    block = dim3(div_up(state.num_total_hash_entries_, thread.x));

    check_visibility_flag_kernel<<<block, thread>>>(
        map_struct,
        flag.get(),
        frame_pose.inverse().cast<float>().matrix3x4(),
        cols, rows,
        K(0,0), K(1,1),
        K(0,2), K(1,2));
    hipDeviceSynchronize();
    safe_call(hipGetLastError());
    printf("checked visibility flag   ");

    thrust::exclusive_scan(flag, flag + state.num_total_hash_entries_, pos_array);

    copy_visible_block_kernel<<<block, thread>>>(
        map_struct.hash_table_,
        visible_blocks,
        flag.get(),
        pos_array.get());
    hipDeviceSynchronize();
    safe_call(hipGetLastError());
    printf("copied visible blocks   ");

    visible_block_count = pos_array[state.num_total_hash_entries_ - 1];

    if (visible_block_count == 0)
        return;

}

void check_visibility(
    MapStorage map_struct,
    MapState state,
    const cv::cuda::GpuMat depth,
    const Sophus::SE3d &frame_pose,
    const Eigen::Matrix3f K,
    cv::cuda::GpuMat &cv_flag,
    cv::cuda::GpuMat &cv_pos_array,
    HashEntry *visible_blocks,
    uint &visible_block_count)
{
    if (cv_flag.empty())
        cv_flag.create(1, state.num_total_hash_entries_, CV_8UC1);
    if (cv_pos_array.empty())
        cv_pos_array.create(1, state.num_total_hash_entries_, CV_32SC1);

    thrust::device_ptr<uchar> flag(cv_flag.ptr<uchar>());
    thrust::device_ptr<int> pos_array(cv_pos_array.ptr<int>());

    const int cols = depth.cols;
    const int rows = depth.rows;

    dim3 thread(MAX_THREAD);
    dim3 block(div_up(state.num_total_hash_entries_, thread.x));

    check_visibility_flag_kernel<<<block, thread>>>(
        map_struct,
        flag.get(),
        frame_pose.inverse().cast<float>().matrix3x4(),
        cols, rows,
        K(0,0), K(1,1),
        K(0,2), K(1,2));
    hipDeviceSynchronize();
    safe_call(hipGetLastError());

    thrust::exclusive_scan(flag, flag + state.num_total_hash_entries_, pos_array);

    copy_visible_block_kernel<<<block, thread>>>(
        map_struct.hash_table_,
        visible_blocks,
        flag.get(),
        pos_array.get());
    hipDeviceSynchronize();
    safe_call(hipGetLastError());

    visible_block_count = pos_array[state.num_total_hash_entries_ - 1];
}

void color_objects(
    MapStorage map_struct,
    MapState state,
    const cv::cuda::GpuMat depth,
    const cv::cuda::GpuMat image,
    const cv::cuda::GpuMat mask,
    const Sophus::SE3d &frame_pose,
    const Eigen::Matrix3f K,
    cv::cuda::GpuMat &cv_flag,
    cv::cuda::GpuMat &cv_pos_array,
    HashEntry *visible_blocks,
    uint &visible_block_count)
{
    if (cv_flag.empty())
        cv_flag.create(1, state.num_total_hash_entries_, CV_8UC1);
    if (cv_pos_array.empty())
        cv_pos_array.create(1, state.num_total_hash_entries_, CV_32SC1);

    thrust::device_ptr<uchar> flag(cv_flag.ptr<uchar>());
    thrust::device_ptr<int> pos_array(cv_pos_array.ptr<int>());

    const int cols = depth.cols;
    const int rows = depth.rows;

    dim3 thread, block;

    // find all visible blocks
    thread = dim3(MAX_THREAD);
    block = dim3(div_up(state.num_total_hash_entries_, thread.x));
    check_visibility_flag_kernel<<<block, thread>>>(
        map_struct,
        flag.get(),
        frame_pose.inverse().cast<float>().matrix3x4(),
        cols, rows,
        K(0,0), K(1,1),
        K(0,2), K(1,2));

    thrust::exclusive_scan(flag, flag + state.num_total_hash_entries_, pos_array);

    copy_visible_block_kernel<<<block, thread>>>(
        map_struct.hash_table_,
        visible_blocks,
        flag.get(),
        pos_array.get());

    visible_block_count = pos_array[state.num_total_hash_entries_ - 1];

    if (visible_block_count == 0)
        return;

    // update object information in the map
    thread = dim3(8, 8);
    block = dim3(visible_block_count);
    update_map_with_object_kernel<<<block, thread>>>(
        map_struct,
        visible_blocks,
        visible_block_count,
        depth,
        image,
        mask,
        frame_pose.inverse().cast<float>().matrix3x4(),
        K(0,0), K(1,1),
        K(0,2), K(1,2));
}

// __global__ void update_cuboids_dimension_kernel(
//     cv::cuda::PtrStep<Vector4f> vmap,
//     cv::cuda::PtrStepSz<unsigned char> mask,
//     cv::cuda::PtrStepSz<float> bbox,
//     unsigned char label,
//     cv::cuda::PtrStepSz<float> &cuboid)
// {
//     float x_min, y_min, x_max, y_max;   // x -> cols, y -> rows
//     x_min = bbox.ptr(0)[0];
//     y_min = bbox.ptr(0)[1];
//     x_max = bbox.ptr(0)[2];
//     y_max = bbox.ptr(0)[3];
//     int v = threadIdx.x;
//     int u = blockIdx.x;

//     // we only care the region inside the bbox
//     if(u<x_min || u>x_max || v<y_min || v>y_max)
//         return;

//     // check label consistency
//     if(mask.ptr(v)[u] != label)
//         return;

//     Vector4f vertex = vmap.ptr(v)[u];
//     float tmp_x = vertex.x / vertex.w;
//     float tmp_y = vertex.y / vertex.w;
//     float tmp_z = vertex.z / vertex.w;


// }

// void estimate_cuboids(
//     const cv::cuda::GpuMat vmap,
//     const cv::cuda::GpuMat mask,
//     const cv::cuda::GpuMat bbox,
//     const unsigned char label,
//     cv::cuda::GpuMat &cuboid)
// {
//     const int cols = vmap.cols;
//     const int rows = vmap.rows;

//     cuboid.create(1, 6. CV_32F);

//     dim3 thread, block;
//     thread = dim3(rows);
//     block = dim3(cols);
//     estimate_cuboids<<<block, thread>>>(
//         vmap,
//         mask,
//         bbox,
//         label,
//         cuboid);

// }


} // namespace cuda
} // namespace fusion